
#include <hip/hip_runtime.h>

#define STRLEN 64

#define LEVENSHTEIN_MAX_LENTH 255

#define LEVDROWSIZE (LEVENSHTEIN_MAX_LENTH+1)

struct levdist_in {
	int l1, l2;
	int r;
	long long int ts, te;
	char s1[STRLEN];
	char s2[STRLEN];
};

struct levdist_out {
	int r;
	long long int ts, te;
};

extern "C" __global__ void levdist (levdist_in *lvd_in, levdist_out *lvd_out)
{
	int *p1, *p2, *tmp;
	int i1, i2, c0, c1, c2;
	int l1 = lvd_in->l1, l2 = lvd_in->l2;

	if (l1==0) { lvd_out[threadIdx.x].r = l2; return; }
	if (l2==0) { lvd_out[threadIdx.x].r = l1; return; }

	if (l1>LEVENSHTEIN_MAX_LENTH || l2>LEVENSHTEIN_MAX_LENTH) {
		lvd_out[threadIdx.x].r = -1;
		return;
	}
	
	int levdp1a [LEVDROWSIZE];
	int levdp2a [LEVDROWSIZE];

	p1 = levdp1a;
	p2 = levdp2a;

	for (i2=0; i2<=l2; i2++)
		p1 [i2] = i2;

	for (i1=0; i1<l1; i1++)
	{
		p2 [0] = p1 [0] + 1;
		for (i2=0; i2<l2; i2++)
		{
			c0 = p1 [i2] + (lvd_in->s1[i1] == lvd_in->s2[i2] ? 0 : 1);
			c1 = p1 [i2+1] + 1;
			if (c1<c0) c0 = c1;
			c2 = p2 [i2] + 1;
			if (c2<c0) c0 = c2;
			p2 [i2+1] = c0;
		}
		tmp=p1; p1=p2; p2=tmp;
	}

//	c0 = p1 [l2];

	lvd_out [threadIdx.x].r = c0;
}
